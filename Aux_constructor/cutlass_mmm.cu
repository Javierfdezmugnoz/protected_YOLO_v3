#include <iostream>
#include "cutlass_mmm.h"

#include <cutlass/numeric_types.h>
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include "hip/hip_runtime.h"
#include <MMM_protected/MMM_protected.h>

__device__ __constant__ uint32_t d_CRC_table_constant[CRC_TABLE_ELEMENTS];

void aux_funct(void){
  printf("Hello cutlass_mmm library\n");
  return;
}

//Define a CUTLASS GEMM template and launch a GEMM kernel.
//hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c) 
hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c) 
{
  // Define type definition for single-precision CUTLASS GEMM with column-major
  // input matrices and 128x128x8 threadblock tile size (chosen by default).
  //
  // To keep the interface manageable, several helpers are defined for plausible compositions
  // including the following example for single-precision GEMM. Typical values are used as
  // default template arguments. See `cutlass/gemm/device/default_gemm_configuration.h` for more details.
  //
  // To view the full gemm device API interface, see `cutlass/gemm/device/gemm.h`
  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  // std::cout << M << "\t N:" << N << "\t K:" << K << "\t A:" << A[0] << "\t B:" << B[0] << "\t C:" << C[0] << "\t alpha:" << alpha << "\t beta:" << beta << "\t" << "d_ES_a[0]" << "\t" << "d_ES_b[0]" << "\t" <<" d_ES_c[0]" << "\n" ;
  // printf("M=%d\t N=%d\t K=%d\tEXTERNAL_ES:%d\n",M,N,K,EXTERNAL_ES);

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                          ColumnMajor,  // Layout of A matrix
                                          float,        // Data-type of B matrix
                                          ColumnMajor,  // Layout of B matrix
                                          float,        // Data-type of C matrix
                                          ColumnMajor>; // Layout of C matrix

  // // Define a CUTLASS GEMM type
   CutlassGemm gemm_operator;

  //printf("\n Direction of h_ES_0: %p and value: %f \n", (void *) h_ES_0, h_ES_0[4]);

  // Construct the CUTLASS GEMM arguments object.
  //
  // One of CUTLASS's design patterns is to define gemm argument objects that are constructible
  // in host code and passed to kernels by value. These may include pointers, strides, scalars,
  // and other arguments needed by Gemm and its components.
  //
  // The benefits of this pattern are (1.) a structured, composable strategy for passing host-constructible
  // arguments to kernels and (2.) minimized initialization overhead on kernel entry.
  //
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source IImatrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}// Scalars used in the Epilogue
                              ,d_ES_a,      // Pointer to d_ES_a
                              d_ES_b,      // Pointer to d_ES_b
                              d_ES_c
                              );     // Pointer to d_ES_c
                             


  // ===============================================
  //        Launch the CUTLASS GEMM kernel.
  // ===============================================
  // hipDeviceSynchronize();
  cutlass::Status status = gemm_operator(args);
  // hipDeviceSynchronize();
  // printf("Status error:%s\n",cutlassGetStatusString(status));

  // ===============================================
  //    Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  // ===============================================


  if (status != cutlass::Status::kSuccess) {
    printf("Status error:%s\n",cutlassGetStatusString(status));
    return hipErrorUnknown;
  }
  // Return success, if no errors were encountered.
  return hipSuccess;
}

hipError_t CutlassSgemmNT(int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c, hipStream_t stream) 
{
  // Define type definition for single-precision CUTLASS GEMM with column-major
  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  // printf("M=%d\t N=%d\t K=%d\t A=%p\t B=%p\t C=%p\n",M,N,K,A,B,C);

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                          RowMajor,     // Layout of A matrix
                                          float,        // Data-type of B matrix
                                          ColumnMajor,  // Layout of B matrix
                                          float,        // Data-type of C matrix
                                          ColumnMajor>; // Layout of C matrix

  // ===============================================
  //      Define a CUTLASS GEMM type
  // ===============================================
   CutlassGemm gemm_operator;

  // ===============================================
  // Construct the CUTLASS GEMM arguments object.
  // ===============================================
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source IImatrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}// Scalars used in the Epilogue
                              ,d_ES_a,      // Pointer to d_ES_a
                              d_ES_b,      // Pointer to d_ES_b
                              d_ES_c
                              );     // Pointer to d_ES_c
                             
  // ===============================================
  //        Launch the CUTLASS GEMM kernel.
  // ===============================================
  hipDeviceSynchronize();
  cutlass::Status status = gemm_operator(args);
  hipDeviceSynchronize();
  printf("Status error:%s\n",status);

  // ===============================================
  //    Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  // ===============================================
  if (status != cutlass::Status::kSuccess) {
    printf("Status error:%s\n",status);
    return hipErrorUnknown;
  }
  // Return success, if no errors were encountered.
  return hipSuccess;
}

hipError_t CutlassSgemmTT(int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c, hipStream_t stream) 
{
  // Define type definition for single-precision CUTLASS GEMM with column-major
  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  // printf("M=%d\t N=%d\t K=%d\t A=%p\t B=%p\t C=%p\n",M,N,K,A,B,C);

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                          RowMajor,     // Layout of A matrix
                                          float,        // Data-type of B matrix
                                          RowMajor,     // Layout of B matrix
                                          float,        // Data-type of C matrix
                                          ColumnMajor>; // Layout of C matrix

  // ===============================================
  //      Define a CUTLASS GEMM type
  // ===============================================
   CutlassGemm gemm_operator;

  // ===============================================
  // Construct the CUTLASS GEMM arguments object.
  // ===============================================
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source IImatrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}// Scalars used in the Epilogue
                              ,d_ES_a,      // Pointer to d_ES_a
                              d_ES_b,      // Pointer to d_ES_b
                              d_ES_c
                              );     // Pointer to d_ES_c
                             
  // ===============================================
  //        Launch the CUTLASS GEMM kernel.
  // ===============================================
  hipDeviceSynchronize();
  cutlass::Status status = gemm_operator(args);
  hipDeviceSynchronize();
  printf("Status error:%s\n",status);

  // ===============================================
  //    Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  // ===============================================
  if (status != cutlass::Status::kSuccess) {
    printf("Status error:%s\n",status);
    return hipErrorUnknown;
  }
  // Return success, if no errors were encountered.
  return hipSuccess;
}

hipError_t CutlassSgemmTN(int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c, hipStream_t stream) 
{
  // Define type definition for single-precision CUTLASS GEMM with column-major
  using ColumnMajor = cutlass::layout::ColumnMajor;
  using RowMajor = cutlass::layout::RowMajor;
  // printf("M=%d\t N=%d\t K=%d\t A=%p\t B=%p\t C=%p\n",M,N,K,A,B,C);

  using CutlassGemm = cutlass::gemm::device::Gemm<float,        // Data-type of A matrix
                                          ColumnMajor,  // Layout of A matrix
                                          float,        // Data-type of B matrix
                                          RowMajor,  // Layout of B matrix
                                          float,        // Data-type of C matrix
                                          ColumnMajor>; // Layout of C matrix

  // ===============================================
  //      Define a CUTLASS GEMM type
  // ===============================================
   CutlassGemm gemm_operator;

  // ===============================================
  // Construct the CUTLASS GEMM arguments object.
  // ===============================================
  CutlassGemm::Arguments args({M , N, K},  // Gemm Problem dimensions
                              {A, lda},    // Tensor-ref for source matrix A
                              {B, ldb},    // Tensor-ref for source IImatrix B
                              {C, ldc},    // Tensor-ref for source matrix C
                              {C, ldc},    // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                              {alpha, beta}// Scalars used in the Epilogue
                              ,d_ES_a,      // Pointer to d_ES_a
                              d_ES_b,      // Pointer to d_ES_b
                              d_ES_c
                              );     // Pointer to d_ES_c
                             
  // ===============================================
  //        Launch the CUTLASS GEMM kernel.
  // ===============================================
  hipDeviceSynchronize();
  cutlass::Status status = gemm_operator(args);
  hipDeviceSynchronize();
  printf("Status error:%s\n",status);

  // ===============================================
  //    Return a hipError_t if the CUTLASS GEMM operator returned an error code.
  // ===============================================
  if (status != cutlass::Status::kSuccess) {
    printf("Status error:%s\n",status);
    return hipErrorUnknown;
  }
  // Return success, if no errors were encountered.
  return hipSuccess;
}


hipError_t CutlassSgemm(int TA, int TB, int M, int N, int K, float alpha, float *A, int lda, float *B, int ldb, float beta, float *C, int ldc, uint32_t *d_ES_a, uint32_t *d_ES_b, uint32_t *d_ES_c, hipStream_t stream) 
{

  if(!TA && !TB){
    // printf("NN\n");
    return CutlassSgemmNN(M,N,K,alpha,A,lda,B,ldb,beta,C,ldc, d_ES_a, d_ES_b, d_ES_c);
	}
	else if(TA && !TB){
    // printf("TN\n");
	  return CutlassSgemmTN(M,N,K,alpha,A,lda,B,ldb,beta,C,ldc, d_ES_a, d_ES_b, d_ES_c, nullptr);
	}
	else if(!TA && TB){
    // printf("NT\n");
    return CutlassSgemmNT(M,N,K,alpha,A,lda,B,ldb,beta,C,ldc, d_ES_a, d_ES_b, d_ES_c, nullptr);
  }
	else{
    // printf("TT\n");
    return CutlassSgemmTT(M,N,K,alpha,A,lda,B,ldb,beta,C,ldc, d_ES_a, d_ES_b, d_ES_c, nullptr);
  }
}