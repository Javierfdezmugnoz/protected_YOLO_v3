#include "hip/hip_runtime.h"
// Standard Library includes
#pragma once
#include <iostream>
#include <sstream>
#include <vector>

#include <sched.h>
#include <string.h>

// Defines cutlass::gemm::device::Gemm, the generic Gemm computation template class.
#include "MMM_protected.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
//
// This function defines a CUTLASS GEMM kernel instantiation, constructs its parameters object,
// and launches it on the CUDA device.
//
///////////////////////////////////////////////////////////////////////////////////////////////////
#include "hip/hip_runtime.h"


/* ==========================================================================
  Descritption: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) adding
  the carry bit. After that, one's complement is implemented (bit negation). 
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf 
=============================================================================*/
//  __device__ uint32_t _xor (uint32_t ui32_a, uint32_t ui32_b)
// {
//     uint32_t acc;
//     asm ("xor.b32  %0, %1, %2;\n\t"
//                 : "=r"(acc)
//                 : "r"(ui32_a), "r"(ui32_b));
//     return acc;
// }



/* ==========================================================================
  Descritption: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) adding
  the carry bit. After that, one's complement is implemented (bit negation). 
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf 
=============================================================================*/
// __device__ uint32_t __a1c (uint32_t ui32_a, uint32_t ui32_b)
// {
//     uint32_t acc;
//     asm ("add.cc.u32      %0, %1, %2;\n\t"
//          "addc.u32        %0, %0, 0;\n\t"
//          "not.b32         %0, %0;\n\t"
//          : "=r"(acc)
//          : "r"(ui32_a), "r"(ui32_b));
//     return acc;
// }


/* ==========================================================================
  Descritption: Ones complement with atomic instructions
=============================================================================*/
__device__ uint32_t a1c_atomic (uint32_t ui32_a, uint32_t ui32_b)
{
    atomicAdd((uint32_t*) &ui32_a, ui32_b);
    ui32_a = ~ui32_a;
    return ui32_a;
}


/* ==========================================================================
  Description: Addition of two values using PTX (parallel thread execution) 
  and ISA (parallel thread execution with instruction set architecture) with
  not carry-bit addition. This operation is followed by the two's complement
  implementation (bit negation and then, addition of 1)
  Additional info:
  url: https://docs.nvidia.com/cuda/parallel-thread-execution/index.html or
  https://docs.nvidia.com/pdf/ptx_isa_5.0.pdf
=============================================================================*/
// __device__ uint32_t __a2c (uint32_t ui32_a, uint32_t ui32_b)
// {
//     uint32_t acc = 0;
//     asm ("add.u32     %0, %1, %2;\n\t"
//          "not.b32     %0, %0;\n\t"
//          "add.u32     %0, %0, 1;\n\t"
//          : "=r"(acc)
//          : "r"(ui32_a), "r"(ui32_b));
//     return acc;
// }

/* ==========================================================================
  Descritption: Two's complement with atomic instructions
=============================================================================*/
__device__ uint32_t a2c_atomic (uint32_t ui32_a, uint32_t ui32_b)
{
    atomicAdd((uint32_t*) &ui32_a, ui32_b);
    ui32_a = ~ui32_a;
    atomicAdd((uint32_t*) &ui32_a, 1u);
    return ui32_a;
}

/* ==========================================================================
  Description: CRC checksum
=============================================================================*/
// __device__  uint32_t singletable_crc32c_ui32(uint32_t *d_CRC_table_shared, uint32_t ui32_crc, uint32_t ui32_data)
//   {
//     ui32_to_ui8_t u;
//     u.ui32 = ui32_data;
//     /* 4 bytes*/
//     ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[0u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[1u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[2u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = d_CRC_table_shared[(ui32_crc ^ u.ui8[3u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     return ui32_crc;
//   }

// __host__ uint32_t h_singletable_crc32c_ui32(uint32_t *kaui32_crc_table, uint32_t ui32_crc, uint32_t ui32_data)
//   {
//     ui32_to_ui8_t u;
//     u.ui32 = ui32_data;

//     /* 4 bytes*/
//     ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[0u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[1u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[2u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     ui32_crc = kaui32_crc_table[(ui32_crc ^ u.ui8[3u]) & 0x00ffu] ^ (ui32_crc >> 8u);
//     return ui32_crc;
//   }

/* ==========================================================================
  Description: Fletcher checksum
=============================================================================*/
// __device__ uint32_t Fletcher32c_ui32(uint32_t Prev_Fletcher, uint32_t ui32_data)
// {
// 	  ui32_to_ui16_t v;
//     ui32_to_ui16_t Fletcher;
//     v.ui32        = ui32_data;
//     Fletcher.ui32 = Prev_Fletcher;

// 	Fletcher.ui16[0] += v.ui16[0];
// 	Fletcher.ui16[1] += Fletcher.ui16[0];
// 	Fletcher.ui16[0] += v.ui16[1];
// 	Fletcher.ui16[1] += Fletcher.ui16[0];
//   /*
//   Fletcher.ui16[0] = Fletcher.ui16[0] % 255
//   Fletcher.ui16[0] = Fletcher.ui16[0] & (FLET_DIVISOR-1)
//   */

// 	Fletcher.ui16[0] %= 255;
// 	Fletcher.ui16[1] %= 255;

// 	return Fletcher.ui32;
// }


// ==========================================================================
//  Descritption: Two's complement with atomic instructions
// ==========================================================================
//  __device__ uint32_t get_smid(void) 
// {
//      uint32_t ret;
//      asm("mov.u32 %0, %%smid;" : "=r"(ret) );
//      return ret;
// }